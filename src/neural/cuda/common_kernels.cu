#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.

  Additional permission under GNU GPL version 3 section 7

  If you modify this Program, or any covered work, by linking or
  combining it with NVIDIA Corporation's libraries from the NVIDIA CUDA
  Toolkit and the NVIDIA CUDA Deep Neural Network library (or a
  modified version of those libraries), containing parts covered by the
  terms of the respective license agreement, the licensors of this
  Program grant you additional permission to convey the resulting work.
*/

#include "cuda_common.h"
#include "neural/network.h"

namespace lczero {
namespace cudnn_backend {

/////////////////////////////////////////////////////////////////////////////
//          Simple CUDA kernels used by certain layers                     //
/////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void addVectors_kernel(T* c, T* a, T* b, int size, int asize,
                                  int bsize, bool relu, bool useTanh,
                                  bool useSigmoid) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    float aVal = 0;
    float bVal = 0;
    if (a) aVal = (float)(a[i % asize]);
    if (b) bVal = (float)(b[i % bsize]);

    float cVal = aVal + bVal;

    if (relu && (cVal < 0)) cVal = 0;

    if (useTanh) {
      cVal = tanh(cVal);
    }

    if (useSigmoid) {
      cVal = 1.0f / (1.0f + exp(-cVal));
    }

    c[i] = (T)cVal;
  }
}

// Adds two vectors (possibly of different sizes), also do optional relu
// activation.
template <typename T>
void addVectors(T* c, T* a, T* b, int size, int asize, int bsize, bool relu,
                bool use_tanh, bool use_sigmoid) {
  const int kBlockSize = 256;
  int blocks = DivUp(size, kBlockSize);

  addVectors_kernel<<<blocks, kBlockSize>>>(c, a, b, size, asize, bsize, relu,
                                            use_tanh, use_sigmoid);
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void addBias_NCHW_kernel(T* c, T* a, T* b, int N, int C, int H,
                                    int W) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int size = N * C * H * W;
  if (i < size) {
    float aVal = (float)a[i];

    // All this math can be optimized, but the kernel is memory bound anyway
    int biasIndex = (i / (H * W)) % C;
    float bVal = (float)b[biasIndex];

    float cVal = aVal + bVal;
    c[i] = (T)cVal;
  }
}

// add bias to convolution's output
template <typename T>
void addBias_NCHW(T* c, T* a, T* b, int N, int C, int H, int W) {
  int size = N * C * H * W;
  const int kBlockSize = 256;
  int blocks = DivUp(size, kBlockSize);

  addBias_NCHW_kernel<<<blocks, kBlockSize>>>(c, a, b, N, C, H, W);
  ReportCUDAErrors(hipGetLastError());
}

__device__ half readNCHW(float* input_tensor, int n, int c, int h, int w,
                         int Nin, int Cin, int H, int W) {
  if (n >= Nin || c >= Cin) return 0;

  int index;
  index = n;
  index *= Cin;
  index += c;
  index *= H;
  index += h;
  index *= W;
  index += w;

  return (half)(input_tensor[index]);
}

__global__ void fp32NCHWtofp16NHWC_kernel(half* output_tensor,
                                          float* input_tensor, int Nin, int Cin,
                                          int Nout, int Cout, int H, int W) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= Nout * Cout * H * W) return;

  int index = tid;

  int c = (index % Cout);
  index /= Cout;
  int w = index % W;
  index /= W;
  int h = index % H;
  index /= H;
  int n = index;

  output_tensor[tid] = readNCHW(input_tensor, n, c, h, w, Nin, Cin, H, W);
}

void fp32NCHWtofp16NHWC(half* output_tensor, float* input_tensor, int Nin,
                        int Cin, int Nout, int Cout, int H, int W) {
  size_t numElements = Nout * Cout * H * W;
  const int blockSize = 256;
  int blocks = DivUp(numElements, blockSize);
  fp32NCHWtofp16NHWC_kernel<<<blocks, blockSize>>>(output_tensor, input_tensor,
                                                   Nin, Cin, Nout, Cout, H, W);
}

template <typename DstType, typename SrcType>
__global__ void copyTypeConverted_kernel(DstType* op, SrcType* ip, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= N) return;

  DstType el = (DstType)ip[tid];
  op[tid] = el;
}

template <typename DstType, typename SrcType>
void copyTypeConverted(DstType* op, SrcType* ip, int N) {
  const int kBlockSize = 256;
  int blocks = DivUp(N, kBlockSize);
  copyTypeConverted_kernel<<<blocks, kBlockSize>>>(op, ip, N);
}

template <typename T>
__global__ void batchNorm_kernel(T* output, const T* input, const T* skipInput,
                                 int N, int C, int H, int W, const float* means,
                                 const float* varMultipliers, bool relu) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int wIndex = 0;
  if (sizeof(T) == sizeof(float))
    wIndex = (index / (H * W)) % C;  // NCHW for fp32
  else
    wIndex = index % C;  // NHWC for fp16

  float el = input[index];
  float mean = means[wIndex];
  float varMulti = varMultipliers[wIndex];

  el -= mean;
  el *= varMulti;

  if (skipInput) el += (float)skipInput[index];

  if (relu && (el < 0)) el = 0;

  output[index] = (T)el;
}

// Every thread processes single element.
template <typename T>
void batchNorm(T* output, const T* input, const T* skipInput, int N, int C,
               int H, int W, float* means, float* var_multipliers, bool relu) {
  const int total_elements = N * C * H * W;
  const int kBlockSize = 256;
  int blocks = DivUp(total_elements, kBlockSize);

  batchNorm_kernel<<<blocks, kBlockSize>>>(output, input, skipInput, N, C, H, W,
                                           means, var_multipliers, relu);

  ReportCUDAErrors(hipGetLastError());
}

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  // Block size of 256, same mask/val for 64 consecutive threads.
  constexpr int kNumShmemElments = 256 / 64;

  __shared__ uint64_t shMasks[kNumShmemElments];
  __shared__ float shVals[kNumShmemElments];

  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int planeIndex = index >> 6;

  if (planeIndex >= n) return;

  // load inputs to shared memory.
  if (threadIdx.x < kNumShmemElments) {
    shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
    shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
  }
  __syncthreads();

  uint64_t mask = shMasks[threadIdx.x >> 6];

  int sqIndex = index & 0x3F;
  float op = 0;

  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    op = shVals[threadIdx.x >> 6];
  }
  output[index] = op;
}

void expandPlanes_Fp32_NCHW(float* output, const uint64_t* masks,
                            const float* values, int n) {
  int threads = n * 8 * 8;  // Each thread writes a single element.
  const int blockSize = 256;
  int blocks = DivUp(threads, blockSize);
  expandPlanes_kernel_Fp32_NCHW<<<blocks, blockSize>>>(output, masks, values,
                                                       n);
  ReportCUDAErrors(hipGetLastError());
}

// TODO: Can optimize using shared memory if this becomes a bottleneck.
__global__ void expandPlanes_kernel_Fp16_NHWC(half* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  const int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index >= n * 8 * 8) return;

  const int planeIndex = index % kInputPlanes;
  const int boardIndex = index / (kInputPlanes * 8 * 8);
  const int sqIndex = (index / kInputPlanes) & 0x3F;

  uint64_t mask = masks[boardIndex * kInputPlanes + planeIndex];

  half op = 0;
  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    float val = values[boardIndex * kInputPlanes + planeIndex];
    op = (half)val;
  }
  output[index] = op;
}

void expandPlanes_Fp16_NHWC(half* output, const uint64_t* masks,
                            const float* values, int n) {
  int threads = n * 8 * 8;  // Each thread writes a single element.
  const int kBlockSize = 256;
  int blocks = DivUp(threads, kBlockSize);
  expandPlanes_kernel_Fp16_NHWC<<<blocks, kBlockSize>>>(output, masks, values,
                                                        n);
  ReportCUDAErrors(hipGetLastError());
}

// Template instantiation
template void copyTypeConverted<half, float>(half* op, float* ip, int N);
template void copyTypeConverted<float, half>(float* op, half* ip, int N);

template void batchNorm<float>(float* output, const float* input,
                               const float* skipInput, int N, int C, int H,
                               int W, float* means, float* var_multipliers,
                               bool relu);
template void batchNorm<half>(half* output, const half* input,
                              const half* skipInput, int N, int C, int H, int W,
                              float* means, float* var_multipliers, bool relu);

template void addVectors<float>(float* c, float* a, float* b, int size,
                                int asize, int bsize, bool relu, bool use_tanh,
                                bool use_sigmoid);
template void addVectors<half>(half* c, half* a, half* b, int size, int asize,
                               int bsize, bool relu, bool use_tanh,
                               bool use_sigmoid);

template void addBias_NCHW<float>(float* c, float* a, float* b, int N, int C,
                                  int H, int W);

template void addBias_NCHW<half>(half* c, half* a, half* b, int N, int C,
                                  int H, int W);

}  // namespace cudnn_backend
}  // namespace lczero
